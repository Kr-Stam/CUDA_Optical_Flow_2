#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <>

//TODO: remove cpu later
#include "OptFlowCpu.hpp"
#include "OptFlowUtils.hpp"
#include "kernels.hpp"

namespace gpu 
{

	/**
	 * \brief CUDA kernel that creates a grayscale image using the average rgb values, each block is a line
	 *
	 * \param src Source Image
	 * \param dest Destination
	 * \param w Width
	 * \param h Height
	 *
	 * \details IMPORTANT: This kernel should be called with a 1D block, each block is one line of the image
	 */
	__global__ void g_grayscale_avg_1d(const unsigned char *src, unsigned char *dest, int w, int h)
	{
			int x = threadIdx.x;
			int y = blockIdx.x;

			if (x >= w || y >= h)
			{
					return;
			}

			int pos = (y * w + x) * 3;
			int avg = (src[pos] + src[pos + 1] + src[pos + 2]) / 3;
			dest[pos] = dest[pos + 1] = dest[pos + 2] = (unsigned char)avg;
	}

	/**
	 * \brief CUDA kernel that creates a grayscale image using the average rgb values, each block is a rectangle
	 *
	 * \param src Source Image
	 * \param dest Destination
	 * \param w Width
	 * \param h Height
	 *
	 * \details IMPORTANT: This kernel should be called with a 2D block, each block is a square of the image
	 */
	__global__ void g_grayscale_avg_2d(const unsigned char *src, unsigned char *dest, int w, int h)
	{
			int x = threadIdx.x + blockIdx.x * blockDim.x;
			int y = blockIdx.y * blockDim.y + threadIdx.y;

			if (x >= w || y >= h)
			{
					return;
			}

			int pos = (y * w + x) * 3;
			int avg = (src[pos] + src[pos + 1] + src[pos + 2]) / 3;
			dest[pos] = dest[pos + 1] = dest[pos + 2] = (unsigned char)avg;
	}

	/**
	 * \brief Launches a CUDA kernel to grayscale an image
	 *
	 * \param srch Source Image
	 * \param dest_h Destination
	 * \param w Width
	 * \param h Height
	 *
	 */
	void grayscale_avg(const unsigned char *src_h, unsigned char *dest_h, int h, int w)
	{

			unsigned char *src_d;
			unsigned char *dest_d;

			size_t size = h * w * 3 * sizeof(unsigned char);

			hipMalloc((void **)&src_d, size);
			hipMalloc((void **)&dest_d, size);

			hipMemcpy(src_d, src_h, size, hipMemcpyHostToDevice);

			int NUM_OF_THREADS = 32;
			dim3 block_size = dim3(NUM_OF_THREADS, NUM_OF_THREADS);
			int GRID_SIZE_X = (int)ceil((float)w / NUM_OF_THREADS);
			int GRID_SIZE_Y = (int)ceil((float)h / NUM_OF_THREADS);
			dim3 grid_size(GRID_SIZE_X, GRID_SIZE_Y);
			g_grayscale_avg_2d<<<grid_size, block_size>>>(src_d, dest_d, w, h);

			hipMemcpy(dest_h, dest_d, size, hipMemcpyDeviceToHost);

			hipFree(dest_d);
			hipFree(src_d);
	}

	/**
	 * \brief Unoptimized CUDA kernel for 2D convolution
	 *
	 * \param src Source Matrix
	 * \param mask Mask Matrix
	 * \param dest Destination Matrix
	 * \param w Width
	 * \param h Heigth
	 * \param mw Mask Width
	 * \param mh Mask Height
	 */
	 __global__ void g_conv_3ch_2d(const unsigned char *src, const float *mask, unsigned char *dest, int w, int h, int mw, int mh)
	{
			int x = threadIdx.x + blockDim.x * blockIdx.x;
			int y = threadIdx.y + blockDim.y * blockIdx.y;

			if (x >= w || y >= h)
			{
					return;
			}

			int pos = y * w + x;

			int tmp[3] = {0, 0, 0};

			int hmw = mw >> 1;
			int hmh = mh >> 1;
			int start_x = x - hmw;
			int start_y = y - hmh;
			int tmp_pos, mask_pos, tmp_x, tmp_y;

			for (int i = 0; i < mh; i++)
			{
					for (int j = 0; j < mw; j++)
					{
							tmp_x = start_x + j;
							tmp_y = start_y + i;
							if (tmp_x >= 0 && tmp_x < w && tmp_y >= 0 && tmp_y < h)
							{
									tmp_pos = tmp_y * w + tmp_x;
									mask_pos = i * mw + j;
									tmp[0] += src[tmp_pos * 3] * mask[mask_pos];
									tmp[1] += src[tmp_pos * 3 + 1] * mask[mask_pos];
									tmp[2] += src[tmp_pos * 3 + 2] * mask[mask_pos];
							}
					}
			}
			dest[pos * 3] = (unsigned char)tmp[0];
			dest[pos * 3 + 1] = (unsigned char)tmp[1];
			dest[pos * 3 + 2] = (unsigned char)tmp[2];
	}

	/**
	 * \brief Launch a CUDA kernel to perform 2D convolution
	 *
	 * \param src Source Matrix
	 * \param dest Destination Matrix
	 * \param w Width
	 * \param h Height
	 * \param mask_t Mask Matrix
	 * \param mw Mask Width <=5
	 * \param mh Mask Height <=5
	 */
	void conv_3ch_2d(const unsigned char *src_h, unsigned char *dest_h, int w, int h, const float *mask_t, int mw, int mh)
	{

			size_t size = w * h * 3 * sizeof(unsigned char);

			unsigned char *src_d;
			unsigned char *dest_d;
			float *mask_d;

			hipMalloc((void **)&src_d, size);
			hipMalloc((void **)&dest_d, size);
			hipMalloc((void **)&mask_d, mw * mh * sizeof(float));

			hipMemcpy(src_d, src_h, size, hipMemcpyHostToDevice);
			hipMemcpy(mask_d, mask_t, mw * mh * sizeof(float), hipMemcpyHostToDevice);

			int NUM_OF_THREADS = 32;
			dim3 blockSize(NUM_OF_THREADS, NUM_OF_THREADS);
			int GRID_SIZE_X = (int)ceil((float)w / NUM_OF_THREADS);
			int GRID_SIZE_Y = (int)ceil((float)h / NUM_OF_THREADS);
			dim3 gridSize(GRID_SIZE_X, GRID_SIZE_Y);
			g_conv_3ch_2d<<<blockSize, gridSize>>>(src_d, mask_d, dest_d, w, h, mw, mh);

			hipMemcpy(dest_h, dest_d, size, hipMemcpyDeviceToHost);

			hipFree(src_d);
			hipFree(dest_d);
			hipFree(mask_d);
	}

	__constant__ float mask[25];

	/**
	 * \brief A more optimized 2D convolution where the mask is loaded into constant GPU memory before execution
	 *
	 * \param src Source Matrix
	 * \param dest Destination Matrix
	 * \param w Width
	 * \param h Height
	 * \param mw Mask Width
	 * \param mh Mask Height
	 */ 
	__global__ void g_conv_3ch_2d_constant(const unsigned char *src, unsigned char *dest, int w, int h, int mw, int mh)
	{

			int x = threadIdx.x + blockDim.x * blockIdx.x;
			int y = threadIdx.y + blockDim.y * blockIdx.y;

			if (x >= w || y >= h)
			{
					return;
			}

			int pos = y * w + x;

			int tmp[3] = {0, 0, 0};

			int hmw = mw >> 1;
			int hmh = mh >> 1;
			int start_x = x - hmw;
			int start_y = y - hmh;
			int tmp_pos, mask_pos, tmp_x, tmp_y;

			for (int i = 0; i < mh; i++)
			{
					for (int j = 0; j < mw; j++)
					{
							tmp_x = start_x + j;
							tmp_y = start_y + i;
							if (tmp_x >= 0 && tmp_x < w && tmp_y >= 0 && tmp_y < h)
							{
									tmp_pos = tmp_y * w + tmp_x;
									mask_pos = i * mw + j;
									tmp[0] += src[tmp_pos * 3] * mask[mask_pos];
									tmp[1] += src[tmp_pos * 3 + 1] * mask[mask_pos];
									tmp[2] += src[tmp_pos * 3 + 2] * mask[mask_pos];
							}
					}
			}
			dest[pos * 3] = (unsigned char)tmp[0];
			dest[pos * 3 + 1] = (unsigned char)tmp[1];
			dest[pos * 3 + 2] = (unsigned char)tmp[2];
	}

	/**
	 * \brief Launch a CUDA kernel to perform a 2D convolution with constant memory
	 *
	 * \param src Source Matrix
	 * \param dest Destination Matrix
	 * \param w Width
	 * \param h Height
	 * \param mask_t Mask Matrix
	 * \param mw Mask Width <=5
	 * \param mh Mask Height <=5
	 */
	void conv_3ch_2d_constant(const unsigned char *src_h, unsigned char *dest_h, int w, int h, const float *mask_t, int mw, int mh)
	{

			size_t size = w * h * 3 * sizeof(unsigned char);

			unsigned char *src_d;
			unsigned char *dest_d;

			hipMalloc((void **)&src_d, size);
			hipMalloc((void **)&dest_d, size);

			hipMemcpy(src_d, src_h, size, hipMemcpyHostToDevice);
			hipMemcpyToSymbol(HIP_SYMBOL(mask), mask_t, mw * mh * sizeof(float));

			int NUM_OF_THREADS = 32;
			dim3 blockSize(NUM_OF_THREADS, NUM_OF_THREADS);
			int GRID_SIZE_X = (int)ceil((float)w / NUM_OF_THREADS);
			int GRID_SIZE_Y = (int)ceil((float)h / NUM_OF_THREADS);
			dim3 gridSize(GRID_SIZE_X, GRID_SIZE_Y);
			g_conv_3ch_2d_constant<<<blockSize, gridSize>>>(src_d, dest_d, w, h, mw, mh);

			hipMemcpy(dest_h, dest_d, size, hipMemcpyDeviceToHost);

			hipFree(src_d);
			hipFree(dest_d);
	}

	__global__ void g_conv_3ch_tiled(const unsigned char *src, unsigned char *dest, int w, int h, int mw, int mh, int TILE_SIZE_X, int TILE_SIZE_Y){
			//load all data
			//Objasnuvanje za kako raboti, povekje e ova za licna upotreba
			//Se upotrebuva maksimalniot mozhen blockSize shto e 32x32
			//Se loadiraat site vrednosti vnatre vo toj blockSize
			//Se koristi TILE_SIZE shto e 32-mw+1;
			//Za da se loadiraat vrednosti nadvor od src mora da se napravat input indeksi i output indeksi
			//Mapiranjeto na nivo na thread e out(0,0) e na TILE_SIZE, in(0,0) e na BLOCK_SIZE
			//Site threads loadiraat, ama ako threadot e nadvor od TILE_SIZE togash ne e output thread 

			extern __shared__ unsigned char tile[];    

			int hmh = mh >> 1;
			int hmw = mw >> 1;

			int x_o = threadIdx.x + blockIdx.x * TILE_SIZE_X;
			int y_o = threadIdx.y + blockIdx.y * TILE_SIZE_Y;
			int pos_o = x_o + y_o * w; 
			int x_i = x_o - hmw;
			int y_i = y_o - hmh;

			int tile_pos = threadIdx.x + threadIdx.y * blockDim.x;
			if(x_i < 0 || x_i >= w || y_i < 0 || y_i >= h){
					tile[tile_pos * 3] = tile[tile_pos * 3 + 1] = tile[tile_pos * 3 + 2] = 0;
			}else{
					int pos_i = x_i + y_i * w;
					tile[tile_pos * 3] = src[pos_i * 3];
					tile[tile_pos * 3 + 1] = src[pos_i * 3 + 1];
					tile[tile_pos * 3 + 2] = src[pos_i * 3 + 2];
			}

			__syncthreads();

			if(x_o >= w || y_o >= h){
					return;
			}
			if(threadIdx.x >= TILE_SIZE_X || threadIdx.y >= TILE_SIZE_Y){
					return;
			}

			int tmp_x, tmp_y, tmp_pos, mask_pos;
			float tmp[] = {0, 0, 0};
			for(int i = 0; i < mh; i++){
					tmp_y = threadIdx.y + i;
					for(int j = 0; j < mw; j++){
							tmp_x = threadIdx.x + j;
							tmp_pos = tmp_x + tmp_y * blockDim.x;
							mask_pos = j + i * mw;
							tmp[0] += tile[tmp_pos * 3] * mask[mask_pos];
							tmp[1] += tile[tmp_pos * 3 + 1] * mask[mask_pos];
							tmp[2] += tile[tmp_pos * 3 + 2] * mask[mask_pos];
					}
			}
			dest[pos_o * 3] = (unsigned char) tmp[0]; 
			dest[pos_o * 3 + 1] = (unsigned char) tmp[1]; 
			dest[pos_o * 3 + 2] = (unsigned char) tmp[2]; 

			//Tile e indeksiran na nivo na block
			//Odma gi isfrlame site outputs shto se out of bounds na src    
			//
	}

	void conv_3ch_tiled(const unsigned char *src_h, unsigned char *dest_h, int w, int h, const float *mask_t, int mw, int mh)
	{
			size_t size = w * h * 3 * sizeof(unsigned char);

			unsigned char *src_d;
			unsigned char *dest_d;

			hipMalloc((void **)&src_d, size);
			hipMalloc((void **)&dest_d, size);

			hipMemcpy(src_d, src_h, size, hipMemcpyHostToDevice);
			hipMemcpyToSymbol(HIP_SYMBOL(mask), mask_t, mw * mh * sizeof(float));

			int NUM_OF_THREADS = 32;
			int TILE_SIZE_X = NUM_OF_THREADS - mw + 1;
			int TILE_SIZE_Y = NUM_OF_THREADS - mh + 1;
			dim3 blockSize(NUM_OF_THREADS, NUM_OF_THREADS);
			//? Mozhe da se optimizira ova
			int GRID_SIZE_X = (int)ceil((float)w / TILE_SIZE_X);
			int GRID_SIZE_Y = (int)ceil((float)h / TILE_SIZE_Y);
			dim3 gridSize(GRID_SIZE_X, GRID_SIZE_Y);
			g_conv_3ch_tiled<<<gridSize, blockSize, blockSize.x * blockSize.y * sizeof(unsigned char) * 3>>>(src_d, dest_d, w, h, mw, mh, TILE_SIZE_X, TILE_SIZE_Y);

			hipMemcpy(dest_h, dest_d, size, hipMemcpyDeviceToHost);

			hipFree(src_d);
			hipFree(dest_d);
	}

	/// @brief This is CUDA kernel for 2D convolution, reducing the channels from 3 to 1
	/// @param src_h Source Image
	/// @param w Image Width
	/// @param h Image Height
	/// @param dest_h Destination Image
	/// @param mask_t Mask
	/// @param mw Mask Width (<=5)
	__global__ void g_conv_3ch_1ch_constant(unsigned char *src, int w, int h, unsigned char *dest, int mw, int mh)
	{

			int x = threadIdx.x + blockIdx.x * blockDim.x;
			int y = threadIdx.y + blockIdx.y * blockDim.y;

			if (x >= w || y >= h)
			{
					return;
			}

			int pos = y * w + x;

			int hmw = mw >> 1;
			int hmh = mh >> 1;

			int start_x = x - hmw;
			int start_y = y - hmh;

			int tmp_pos, tmp_x, tmp_y, mask_pos;
			int tmp = 0;
			for (int i = 0; i < mh; i++)
			{
					tmp_y = start_y + i;
					if (tmp_y < 0 || tmp_y >= h)
					{
							continue;
					}
					for (int j = 0; j < mw; j++)
					{
							tmp_x = start_x + j;
							if (tmp_x < 0 || tmp_x >= w)
							{
									continue;
							}
							tmp_pos = tmp_y * w + tmp_x;
							mask_pos = i * mw + j;
							if (mask[mask_pos] == 0)
							{
									continue;
							}
							tmp += src[tmp_pos * 3] * mask[mask_pos];
					}
			}
			dest[pos] = (unsigned char)tmp;
	}

	/// @brief This is a non-tiled implementation of a 2D convolution that loads the mask into constant memory
	/// @param src_h Source Image, size = w * h * 3
	/// @param w Image Width
	/// @param h Image Height
	/// @param dest_h Destination, size = w * h
	/// @param mask_t Mask
	/// @param mw Mask Width (<=5)
	/// @param mh Mask Height (<=5)
	void conv_3ch_1ch_constant(const unsigned char *src_h, int w, int h, unsigned char *dest_h, const float *mask_t, int mw, int mh)
	{
			unsigned char *src_d;
			unsigned char *dest_d;

			hipMalloc((void **)&src_d, w * h * 3 * sizeof(unsigned char));
			hipMalloc((void **)&dest_d, w * h * sizeof(unsigned char));

			hipMemcpy(src_d, src_h, w * h * 3 * sizeof(unsigned char), hipMemcpyHostToDevice);
			hipMemcpyToSymbol(HIP_SYMBOL(mask), mask_t, mw * mh * sizeof(float));

			int NUM_OF_THREADS = 32;
			dim3 blockSize(NUM_OF_THREADS, NUM_OF_THREADS);
			int GRID_SIZE_X = (int)ceil((float)w / (float)NUM_OF_THREADS);
			int GRID_SIZE_Y = (int)ceil((float)h / (float)NUM_OF_THREADS);
			dim3 gridSize(GRID_SIZE_X, GRID_SIZE_Y);
			g_conv_3ch_1ch_constant<<<gridSize, blockSize>>>(src_d, w, h, dest_d, mw, mh);

			hipDeviceSynchronize();
			hipMemcpy(dest_h, dest_d, w * h * sizeof(unsigned char), hipMemcpyDeviceToHost);

			hipFree(src_d);
			hipFree(dest_d);
	}

	
	// TODO: OVOJ KOD IMA PROBLEMI, NE TREBA DA SE KORISTI
	// Ova e za kolku treba da se napravi padding na SHMEM za da se loadiraat vrednostite potrebni za konvolucija so maska od golemina
	// padding*2 + 1
#define SHMEM_PADDING 2
#define PRESUMED_NUM_OF_THREADS 32
	// #define TILE_SIZE 36;
	// naive implementation
	/// @brief This is a CUDA kernel for a tiled implementation of a 2D convolution where the mask is in constant memory
	/// @details IMPORTANT: This function is hardcoded to be run with a block size of 32x32, it may not work with other blockSizes
	/// @param src_h Source Image, size = w * h * 3
	/// @param w Image Width
	/// @param h Image Height
	/// @param dest_h Destination, size = w * h
	/// @param mask_t Mask
	/// @param mw Mask Width (<=5)
	/// @param mh Mask Height (<=5)
	__global__ void g_conv_3ch_1ch_tiled(unsigned char *src, int w, int h, unsigned char *dest, int mw, int mh)
	{

			//? ne znam dali da bide refaktorirano nadvor vo konstanta
			//? Mislam deka ke go napravi kodot samo pozbunuvachki
			// #define SHMEM_PADDING 2;
			// #define PRESUMED_NUM_OF_THREADS 32;
			// #define TILE_SIZE 36;

			__shared__ float tile[36 * 36]; // mnogu me nervira ova treba da razgledam ubavo kako rabotat konstanti vo c, samo ke gi zamenam site vrednosti direktno
			//TODO: Realno ova treba da bide so dinamichna extern shared memorija

			int global_x = threadIdx.x + blockIdx.x * blockDim.x;
			int global_y = threadIdx.y + blockIdx.y * blockDim.y;
			if (global_x >= w || global_y >= h)
			{
					return;
			}
			int global_pos = global_y * w + global_x;

			int local_x = threadIdx.x + 2;
			int local_y = threadIdx.y + 2;
			int local_pos = local_y * 36 + local_x;

			int hmw = mw >> 1;
			int hmh = mh >> 1;

			// Load data into tile

			tile[local_pos] = src[global_pos * 3];

			int tmp_global_x, tmp_global_y, tmp_local_x, tmp_local_y, tmp_global_pos, tmp_local_pos;
			// Left excess
			if (local_x == 2)
			{
					for (int i = 0; i < hmw; i++)
					{
							tmp_global_x = global_x - i;

							tmp_local_pos = local_pos - i;
							if (tmp_global_x < 0)
							{
									tile[tmp_local_pos] = 0;
									//? Ne znam dali e ova potrebno ama better safe than sorry
							}
							else
							{
									tmp_global_pos = global_pos - i;
									tile[tmp_local_pos] = src[tmp_global_pos * 3];
							}
					}
			}
			// Right excess
			if (local_x == 32 + 2 - 1)
			{
					for (int i = 0; i < hmw; i++)
					{
							tmp_global_x = global_x + i;
							tmp_local_pos = local_pos + i;
							if (tmp_global_x >= w)
							{
									tile[tmp_local_pos] = 0;
									//? Ne znam dali e ova potrebno ama better safe than sorry
							}
							else
							{
									tmp_global_pos = global_pos + i;
									tile[tmp_local_pos] = src[tmp_global_pos * 3];
							}
					}
			}

			// Top excess
			if (local_y == 2)
			{
					for (int i = 0; i < hmw; i++)
					{
							tmp_global_y = global_y - i;
							tmp_local_y = local_y - i;

							tmp_local_pos = tmp_local_y * 36 + local_x;

							if (tmp_global_y < 0)
							{
									tile[tmp_local_pos] = 0;
									//? Ne znam dali e ova potrebno ama better safe than sorry
							}
							else
							{
									tmp_global_pos = tmp_global_y * w + global_x;
									tile[tmp_local_pos] = src[tmp_global_pos * 3];
							}
					}
			}
			// Bottom excess
			if (local_y == 32 + 2 - 1)
			{
					for (int i = 0; i < hmw; i++)
					{
							tmp_global_y = global_y + i;
							tmp_local_y = local_y + i;

							tmp_local_pos = tmp_local_y * 36 + local_x;

							if (tmp_global_y >= h)
							{
									tile[tmp_local_pos] = 0;
									//? Ne znam dali e ova potrebno ama better safe than sorry
							}
							else
							{
									tmp_global_pos = tmp_global_y * w + global_x;
									tile[tmp_local_pos] = src[tmp_global_pos * 3];
							}
					}
			}

			// Corners
			// TL
			if (local_x == 2 && local_y == 2)
			{
					int local_start_y = local_y - 2;
					int global_start_y = global_y - 2;
					int local_start_x = local_x - 2;
					int global_start_x = global_x - 2;
					for (int i = 0; i < 2; i++)
					{
							tmp_local_y = local_start_y + i;
							tmp_global_y = global_start_y + i;
							for (int j = 0; j < 2; j++)
							{
									tmp_global_x = global_start_x + i;
									tmp_local_x = local_start_x + i;
									tmp_local_pos = tmp_local_y * 36 + tmp_local_x;
									if (tmp_global_y < 0 || tmp_global_x < 0)
									{
											tile[tmp_local_pos] = 0;
									}
									else
									{
											tmp_global_pos = tmp_global_y * w + global_x;
											tile[tmp_local_pos] = src[tmp_global_pos * 3];
									}
							}
					}
			}
			// TR
			if (local_x == 32 + 2 - 1 && local_y == 2)
			{
					int local_start_y = local_y - 2;
					int global_start_y = global_y - 2;
					int local_start_x = local_x;
					int global_start_x = global_x;
					for (int i = 0; i < 2; i++)
					{
							tmp_local_y = local_start_y + i;
							tmp_global_y = global_start_y + i;
							for (int j = 0; j < 2; j++)
							{
									tmp_global_x = global_start_x + i;
									tmp_local_x = local_start_x + i;
									tmp_local_pos = tmp_local_y * 36 + tmp_local_x;
									if (tmp_global_y < 0 || tmp_global_x < 0)
									{
											tile[tmp_local_pos] = 0;
									}
									else
									{
											tmp_global_pos = tmp_global_y * w + global_x;
											tile[tmp_local_pos] = src[tmp_global_pos * 3];
									}
							}
					}
			}
			// BL
			if (local_x == 2 && local_y == 32 - 2 + 1)
			{
					int local_start_y = local_y;
					int global_start_y = global_y;
					int local_start_x = local_x - 2;
					int global_start_x = global_x - 2;
					for (int i = 0; i < 2; i++)
					{
							tmp_local_y = local_start_y + i;
							tmp_global_y = global_start_y + i;
							for (int j = 0; j < 2; j++)
							{
									tmp_global_x = global_start_x + i;
									tmp_local_x = local_start_x + i;
									tmp_local_pos = tmp_local_y * 36 + tmp_local_x;
									if (tmp_global_y < 0 || tmp_global_x < 0)
									{
											tile[tmp_local_pos] = 0;
									}
									else
									{
											tmp_global_pos = tmp_global_y * w + global_x;
											tile[tmp_local_pos] = src[tmp_global_pos * 3];
									}
							}
					}
			}
			// BR
			if (local_x == 32 - 2 + 1 && local_y == 32 - 2 + 1)
			{
					int local_start_y = local_y;
					int global_start_y = global_y;
					int local_start_x = local_x;
					int global_start_x = global_x;
					for (int i = 0; i < 2; i++)
					{
							tmp_local_y = local_start_y + i;
							tmp_global_y = global_start_y + i;
							for (int j = 0; j < 2; j++)
							{
									tmp_global_x = global_start_x + i;
									tmp_local_x = local_start_x + i;
									tmp_local_pos = tmp_local_y * 36 + tmp_local_x;
									if (tmp_global_y < 0 || tmp_global_x < 0)
									{
											tile[tmp_local_pos] = 0;
									}
									else
									{
											tmp_global_pos = tmp_global_y * w + global_x;
											tile[tmp_local_pos] = src[tmp_global_pos * 3];
									}
							}
					}
			}
			// Loading finished

			__syncthreads();

			// Now the convolution code
			int local_start_x = local_x - hmw;
			int local_start_y = local_y - hmh;
			int tmp = 0;
			int mask_pos;
			for (int i = 0; i < mh; i++)
			{
					tmp_local_y = local_start_y + i;

					for (int j = 0; j < mw; j++)
					{
							tmp_local_x = local_start_x + j;

							tmp_local_pos = tmp_local_y * 36 + tmp_local_x;
							mask_pos = i * mw + j;
							tmp += tile[tmp_local_pos] * mask[mask_pos];
					}
			}
			dest[global_pos] = (unsigned char)tmp;
	}


	/// @brief This is a tiled implementation of a 2D convolution that loads the mask into constant memory
	/// @param src_h Source Image, size = w * h * 3
	/// @param w Image Width
	/// @param h Image Height
	/// @param dest_h Destination, size = w * h
	/// @param mask_t Mask
	/// @param mw Mask Width (<=5)
	/// @param mh Mask Height (<=5)
	void conv_3ch_1ch_tiled(const unsigned char *src_h, int w, int h, unsigned char *dest_h, const float *mask_t, int mw, int mh)
	{
			unsigned char *src_d;
			unsigned char *dest_d;

			hipMalloc((void **)&src_d, w * h * 3 * sizeof(unsigned char));
			hipMalloc((void **)&dest_d, w * h * sizeof(unsigned char));

			hipMemcpy(src_d, src_h, w * h * 3 * sizeof(unsigned char), hipMemcpyHostToDevice);
			hipMemcpyToSymbol(HIP_SYMBOL(mask), mask_t, mw * mh * sizeof(float));

			// Mora ovaa funkcija da se povika so ovaa golemina na blokovi poradi nachinot na koj e napravena shared memorija
			int NUM_OF_THREADS = 32;
			dim3 blockSize(NUM_OF_THREADS, NUM_OF_THREADS);
			int GRID_SIZE_X = (int)ceil((float)w / (float)NUM_OF_THREADS);
			int GRID_SIZE_Y = (int)ceil((float)h / (float)NUM_OF_THREADS);
			dim3 gridSize(GRID_SIZE_X, GRID_SIZE_Y);
			// convolutionGPU2D_3CH_to_1CH_Tiled<<<gridSize, blockSize>>>(src_d, w, h, dest_d, mw, mh);
			g_conv_3ch_1ch_constant<<<gridSize, blockSize>>>(src_d, w, h, dest_d, mw, mh);

			hipDeviceSynchronize();
			hipMemcpy(dest_h, dest_d, w * h * sizeof(unsigned char), hipMemcpyDeviceToHost);

			hipFree(src_d);
			hipFree(dest_d);
	}

	//TODO: OVOJ KOD IMA PROBLEM, NE SMEE DA SE KORISTI
	/// @brief This is a CUDA kernel for a tiled implementation of a 2D convolution where the mask is in constant memory
	/// @details IMPORTANT: This function is hardcoded to be run with a block size of 32x32, it may not work with other blockSizes
	/// @param src_h Source Image, size = w * h * 3
	/// @param w Image Width
	/// @param h Image Height
	/// @param dest_h Destination, size = w * h
	/// @param mask_t Mask
	/// @param mw Mask Width (<=5)
	/// @param mh Mask Height (<=5)
	__global__ void g_conv_3ch_1ch_tiled_uchar_float(unsigned char *src, int w, int h, float *dest, int mw, int mh)
	{

		//? ne znam dali da bide refaktorirano nadvor vo konstanta
		//? Mislam deka ke go napravi kodot samo pozbunuvachki
		// #define SHMEM_PADDING 2;
		// #define PRESUMED_NUM_OF_THREADS 32;
		// #define TILE_SIZE 36;

		__shared__ float tile[36 * 36]; // mnogu me nervira ova treba da razgledam ubavo kako rabotat konstanti vo c, samo ke gi zamenam site vrednosti direktno

		int global_x = threadIdx.x + blockIdx.x * blockDim.x;
		int global_y = threadIdx.y + blockIdx.y * blockDim.y;
		if (global_x >= w || global_y >= h)
		{
				return;
		}
		int global_pos = global_y * w + global_x;

		int local_x = threadIdx.x + 2;
		int local_y = threadIdx.y + 2;
		int local_pos = local_y * 36 + local_x;

		int hmw = mw >> 1;
		int hmh = mh >> 1;

		// Load data into tile

		tile[local_pos] = src[global_pos * 3];

		int tmp_global_x, tmp_global_y, tmp_local_x, tmp_local_y, tmp_global_pos, tmp_local_pos;
		// Left excess
		if (local_x == 2)
		{
				for (int i = 0; i < hmw; i++)
				{
						tmp_global_x = global_x - i;

						tmp_local_pos = local_pos - i;
						if (tmp_global_x < 0)
						{
								tile[tmp_local_pos] = 0;
								//? Ne znam dali e ova potrebno ama better safe than sorry
						}
						else
						{
								tmp_global_pos = global_pos - i;
								tile[tmp_local_pos] = src[tmp_global_pos * 3];
						}
				}
		}
		// Right excess
		if (local_x == 32 + 2 - 1)
		{
				for (int i = 0; i < hmw; i++)
				{
						tmp_global_x = global_x + i;
						tmp_local_pos = local_pos + i;
						if (tmp_global_x >= w)
						{
								tile[tmp_local_pos] = 0;
								//? Ne znam dali e ova potrebno ama better safe than sorry
						}
						else
						{
								tmp_global_pos = global_pos + i;
								tile[tmp_local_pos] = src[tmp_global_pos * 3];
						}
				}
		}

		// Top excess
		if (local_y == 2)
		{
				for (int i = 0; i < hmw; i++)
				{
						tmp_global_y = global_y - i;
						tmp_local_y = local_y - i;

						tmp_local_pos = tmp_local_y * 36 + local_x;

						if (tmp_global_y < 0)
						{
								tile[tmp_local_pos] = 0;
								//? Ne znam dali e ova potrebno ama better safe than sorry
						}
						else
						{
								tmp_global_pos = tmp_global_y * w + global_x;
								tile[tmp_local_pos] = src[tmp_global_pos * 3];
						}
				}
		}
		// Bottom excess
		if (local_y == 32 + 2 - 1)
		{
				for (int i = 0; i < hmw; i++)
				{
						tmp_global_y = global_y + i;
						tmp_local_y = local_y + i;

						tmp_local_pos = tmp_local_y * 36 + local_x;

						if (tmp_global_y >= h)
						{
								tile[tmp_local_pos] = 0;
								//? Ne znam dali e ova potrebno ama better safe than sorry
						}
						else
						{
								tmp_global_pos = tmp_global_y * w + global_x;
								tile[tmp_local_pos] = src[tmp_global_pos * 3];
						}
				}
		}

		// Corners
		// TL
		if (local_x == 2 && local_y == 2)
		{
				int local_start_y = local_y - 2;
				int global_start_y = global_y - 2;
				int local_start_x = local_x - 2;
				int global_start_x = global_x - 2;
				for (int i = 0; i < 2; i++)
				{
						tmp_local_y = local_start_y + i;
						tmp_global_y = global_start_y + i;
						for (int j = 0; j < 2; j++)
						{
								tmp_global_x = global_start_x + i;
								tmp_local_x = local_start_x + i;
								tmp_local_pos = tmp_local_y * 36 + tmp_local_x;
								if (tmp_global_y < 0 || tmp_global_x < 0)
								{
										tile[tmp_local_pos] = 0;
								}
								else
								{
										tmp_global_pos = tmp_global_y * w + global_x;
										tile[tmp_local_pos] = src[tmp_global_pos * 3];
								}
						}
				}
		}
		// TR
		if (local_x == 32 + 2 - 1 && local_y == 2)
		{
				int local_start_y = local_y - 2;
				int global_start_y = global_y - 2;
				int local_start_x = local_x;
				int global_start_x = global_x;
				for (int i = 0; i < 2; i++)
				{
						tmp_local_y = local_start_y + i;
						tmp_global_y = global_start_y + i;
						for (int j = 0; j < 2; j++)
						{
								tmp_global_x = global_start_x + i;
								tmp_local_x = local_start_x + i;
								tmp_local_pos = tmp_local_y * 36 + tmp_local_x;
								if (tmp_global_y < 0 || tmp_global_x < 0)
								{
										tile[tmp_local_pos] = 0;
								}
								else
								{
										tmp_global_pos = tmp_global_y * w + global_x;
										tile[tmp_local_pos] = src[tmp_global_pos * 3];
								}
						}
				}
		}
		// BL
		if (local_x == 2 && local_y == 32 - 2 + 1)
		{
				int local_start_y = local_y;
				int global_start_y = global_y;
				int local_start_x = local_x - 2;
				int global_start_x = global_x - 2;
				for (int i = 0; i < 2; i++)
				{
						tmp_local_y = local_start_y + i;
						tmp_global_y = global_start_y + i;
						for (int j = 0; j < 2; j++)
						{
								tmp_global_x = global_start_x + i;
								tmp_local_x = local_start_x + i;
								tmp_local_pos = tmp_local_y * 36 + tmp_local_x;
								if (tmp_global_y < 0 || tmp_global_x < 0)
								{
										tile[tmp_local_pos] = 0;
								}
								else
								{
										tmp_global_pos = tmp_global_y * w + global_x;
										tile[tmp_local_pos] = src[tmp_global_pos * 3];
								}
						}
				}
		}
		// BR
		if (local_x == 32 - 2 + 1 && local_y == 32 - 2 + 1)
		{
				int local_start_y = local_y;
				int global_start_y = global_y;
				int local_start_x = local_x;
				int global_start_x = global_x;
				for (int i = 0; i < 2; i++)
				{
						tmp_local_y = local_start_y + i;
						tmp_global_y = global_start_y + i;
						for (int j = 0; j < 2; j++)
						{
								tmp_global_x = global_start_x + i;
								tmp_local_x = local_start_x + i;
								tmp_local_pos = tmp_local_y * 36 + tmp_local_x;
								if (tmp_global_y < 0 || tmp_global_x < 0)
								{
										tile[tmp_local_pos] = 0;
								}
								else
								{
										tmp_global_pos = tmp_global_y * w + global_x;
										tile[tmp_local_pos] = src[tmp_global_pos * 3];
								}
						}
				}
		}
		// Loading finished

		__syncthreads();

		// Now the convolution code
		int local_start_x = local_x - hmw;
		int local_start_y = local_y - hmh;
		float tmp = 0;
		int mask_pos;
		for (int i = 0; i < mh; i++)
		{
				tmp_local_y = local_start_y + i;

				for (int j = 0; j < mw; j++)
				{
						tmp_local_x = local_start_x + j;

						tmp_local_pos = tmp_local_y * 36 + tmp_local_x;
						mask_pos = i * mw + j;
						tmp += (float) tile[tmp_local_pos] * mask[mask_pos];
				}
		}
		dest[global_pos] = tmp;
	}


	/// @brief This is CUDA kernel for 2D convolution, reducing the channels from 3 to 1
	/// @param src_h Source Image
	/// @param w Image Width
	/// @param h Image Height
	/// @param dest_h Destination Image
	/// @param mask_t Mask
	/// @param mw Mask Width (<=5)
	__global__ void g_conv_3ch_1ch_constant_uchar_float(unsigned char *src, int w, int h, float *dest, int mw, int mh)
	{

			int x = threadIdx.x + blockIdx.x * blockDim.x;
			int y = threadIdx.y + blockIdx.y * blockDim.y;

			if (x >= w || y >= h)
			{
					return;
			}

			int pos = y * w + x;

			int hmw = mw >> 1;
			int hmh = mh >> 1;

			int start_x = x - hmw;
			int start_y = y - hmh;

			int tmp_pos, tmp_x, tmp_y, mask_pos;
			float tmp = 0;
			for (int i = 0; i < mh; i++)
			{
					tmp_y = start_y + i;
					if (tmp_y < 0 || tmp_y >= h)
					{
							continue;
					}
					for (int j = 0; j < mw; j++)
					{
							tmp_x = start_x + j;
							if (tmp_x < 0 || tmp_x >= w)
							{
									continue;
							}
							tmp_pos = tmp_y * w + tmp_x;
							mask_pos = i * mw + j;
							if (mask[mask_pos] == 0)
							{
									continue;
							}
							tmp += (float) src[tmp_pos * 3] * mask[mask_pos];
					}
			}

			//!MOzhno e ova da se tunira
			// if(tmp > -10 && tmp < 10){
			//     tmp = 0;
			// }
			dest[pos] = tmp;
	}

	/// @brief This is a tiled implementation of a 2D convolution that loads the mask into constant memory
	/// @param src_h Source Image, size = w * h * 3
	/// @param w Image Width
	/// @param h Image Height
	/// @param dest_h Destination, size = w * h
	/// @param mask_t Mask
	/// @param mw Mask Width (<=5)
	/// @param mh Mask Height (<=5)
	void conv_3ch_1ch_tiled_uchar_float(const unsigned char *src_h, int w, int h, float *dest_h, const float *mask_t, int mw, int mh)
	{
			unsigned char *src_d;
			float *dest_d;

			hipMalloc((void **)&src_d, w * h * 3 * sizeof(unsigned char));
			hipMalloc((void **)&dest_d, w * h * sizeof(float));

			hipMemcpy(src_d, src_h, w * h * 3 * sizeof(unsigned char), hipMemcpyHostToDevice);
			hipMemcpyToSymbol(HIP_SYMBOL(mask), mask_t, mw * mh * sizeof(float));

			// Mora ovaa funkcija da se povika so ovaa golemina na blokovi poradi nachinot na koj e napravena shared memorija
			int NUM_OF_THREADS = 32;
			dim3 blockSize(NUM_OF_THREADS, NUM_OF_THREADS);
			int GRID_SIZE_X = (int)ceil((float)w / (float)NUM_OF_THREADS);
			int GRID_SIZE_Y = (int)ceil((float)h / (float)NUM_OF_THREADS);
			dim3 gridSize(GRID_SIZE_X, GRID_SIZE_Y);
			// convolutionGPU2D_3CH_to_1CH_Tiled<<<gridSize, blockSize>>>(src_d, w, h, dest_d, mw, mh);
			g_conv_3ch_1ch_constant_uchar_float<<<gridSize, blockSize>>>(src_d, w, h, dest_d, mw, mh);

			hipDeviceSynchronize();
			hipMemcpy(dest_h, dest_d, w * h * sizeof(float), hipMemcpyDeviceToHost);

			hipFree(src_d);
			hipFree(dest_d);
	}

	/// @brief An unoptimized CUDA kernel for 1D convolutions
	/// @param src Source Array
	/// @param mask Mask Array
	/// @param dest Destination Array
	/// @param m Array Size
	/// @param n Mask Size
	/// @return void
	__global__ void g_conv_1d_3ch(const unsigned char *src, const float *mask_t, unsigned char *dest, int m, int n)
	{
			int x = threadIdx.x + blockIdx.x * blockDim.x;

			if (x >= m)
			{
					return;
			}

			int r = n >> 1;
			int start = x - r;

			int temp[3] = {0, 0, 0};
			for (int i = 0; i < n; i++)
			{
					if (start + i >= 0 && start + i <= m)
					{
							temp[0] += (float)src[(start + i) * 3] * mask_t[i];
							temp[1] += (float)src[(start + i) * 3 + 1] * mask_t[i];
							temp[2] += (float)src[(start + i) * 3 + 2] * mask_t[i];
					}
			}
			dest[x * 3] = (unsigned char)temp[0];
			dest[x * 3 + 1] = (unsigned char)temp[1];
			dest[x * 3 + 2] = (unsigned char)temp[2];
	}

	/// @brief Launches a CUDA kernel to perform 1D convolution, not used just made for practice
	/// @param src Sourc Array
	/// @param dest Destination Array
	void conv_1d_3ch(unsigned char* src_h, int w, int h, unsigned char* dest_h)
	{

			float test[9] = {0.1, 0.2, 0.3, 0.4, 0.5, 0.4, 0.3, 0.2, 0.1};
			int n = 9;

			size_t size = w * h * 3 * sizeof(unsigned char);

			unsigned char *src_d;
			unsigned char *dest_d;
			float *mask_d;
			hipMalloc((void **)&src_d, size);
			hipMalloc((void **)&dest_d, size);
			hipMalloc((void **)&mask_d, n * sizeof(float));

			hipMemcpy(src_d, src_h, size, hipMemcpyHostToDevice);
			hipMemcpy(mask_d, test, n * sizeof(float), hipMemcpyHostToDevice);

			int NUM_OF_THREADS = 1024;
			int NUM_OF_BLOCKS = (int)ceil((float)(w * h) / 1024.0);
			g_conv_1d_3ch<<<NUM_OF_BLOCKS, NUM_OF_THREADS>>>(src_d, mask_d, dest_d, size, n);

			hipMemcpy(dest_h, dest_d, size, hipMemcpyDeviceToHost);

			hipFree(src_d);
			hipFree(dest_d);
			hipFree(mask_d);
	}

	__constant__ float GAUS_KERNEL_3x3_d[9] = {
			0.0625, 0.125, 0.0625,
			0.125, 0.25, 0.125,
			0.0625, 0.125, 0.0625};

	__global__ void g_gauss_pyramid(const unsigned char *src, int w, int h, unsigned char *dest)
	{

			int x = threadIdx.x + blockIdx.x * blockDim.x;
			int y = threadIdx.y + blockIdx.y * blockDim.y;

			if (x >= w || y >= h)
			{
					return;
			}

			float tmp[3] = {0, 0, 0};
			int start_y = (y << 1) - 1;
			int start_x = (x << 1) - 1;
			for (int p = 0; p < 3; p++)
			{
					for (int q = 0; q < 3; q++)
					{
							int cx = start_x + q;
							int cy = start_y + p;
							if (cx >= 0 && cx < w * 2 && cy >= 0 && cy < h * 2)
							{
									int mask_pos = p * 3 + q;
									int img_pos = (cy * w * 2 + cx) * 3;
									tmp[0] += GAUS_KERNEL_3x3_d[mask_pos] * src[img_pos];
									tmp[1] += GAUS_KERNEL_3x3_d[mask_pos] * src[img_pos + 1];
									tmp[2] += GAUS_KERNEL_3x3_d[mask_pos] * src[img_pos + 2];
							}
					}
			}
			int pos = y * w + x;
			dest[pos * 3] = (unsigned char)tmp[0];
			dest[pos * 3 + 1] = (unsigned char)tmp[1];
			dest[pos * 3 + 2] = (unsigned char)tmp[2];
	}

	//TODO: Refactor properly
	void gauss_pyramid_level(const unsigned char *src_h, int w, int h, unsigned char *dest_h, const float* mask, int mw, int mh)
	{
			unsigned char *src_d;
			unsigned char *dest_d;

			int dw = w << 1;
			int dh = h << 1;
			hipMalloc((void **)&src_d, dw * dh * 3 * sizeof(unsigned char));
			hipMalloc((void **)&dest_d, w * h * 3 * sizeof(unsigned char));

			hipMemcpy(src_d, src_h, dw * dh * 3 * sizeof(unsigned char), hipMemcpyHostToDevice);

			int NUM_OF_THREADS = 32;
			dim3 blockSize(NUM_OF_THREADS, NUM_OF_THREADS);
			int GRID_SIZE_X = (int)ceil((float)w / (float)NUM_OF_THREADS);
			int GRID_SIZE_Y = (int)ceil((float)h / (float)NUM_OF_THREADS);
			dim3 gridSize(GRID_SIZE_X, GRID_SIZE_Y);

			gpu::g_gauss_pyramid<<<blockSize, gridSize>>>(src_d, w, h, dest_d);

			hipMemcpy(dest_h, dest_d, w * h * 3 * sizeof(unsigned char), hipMemcpyDeviceToHost);

			hipFree(src_d);
			hipFree(dest_d);
	}

	//? Mozhebi ke dodadam proverka za dali e dovolno golema slikata vo nivoto za da se koristi GPU ili CPU?
	void gauss_pyramid(unsigned char ** pyramid, int w, int h, int levels, const float* mask, int mw, int mh)
	{

			for (int k = 1; k < levels; k++)
			{
					w = w >> 1;
					h = h >> 1;
					gauss_pyramid_level(pyramid[k - 1], w, h, pyramid[k], mask, mw, mh);
			}
	}

	__global__ void g_srm_3ch_1ch_tiled(unsigned char *arr1, unsigned char *arr2, int *dest, int w, int h, int ww, int wh)
	{

			extern __shared__ unsigned char tile1[];
			extern __shared__ unsigned char tile2[];

			int hwh = wh >> 1;
			int hww = ww >> 1;

			const int TILE_SIZE_X = blockDim.x + (hww << 1);
			const int TILE_SIZE_Y = blockDim.y + (hwh << 1);
			const int TILE_SIZE = TILE_SIZE_X * TILE_SIZE_Y;

			int global_x = threadIdx.x + blockIdx.x * blockDim.x;
			int global_y = threadIdx.y + blockIdx.y * blockDim.y;
			int global_pos = global_y * w + global_x;

			int local_x = threadIdx.x + hww;
			int local_y = threadIdx.y + hwh;
			int local_pos = local_y * TILE_SIZE + local_x;

			// Load all values

			tile1[local_pos] = arr1[global_pos * 3];
			tile2[local_pos] = arr2[global_pos * 3];

			// Load excess
			int tmp_global_x, tmp_global_y, tmp_local_x, tmp_local_y, tmp_global_pos, tmp_local_pos;
			// Left excess
			if (local_x == 2)
			{
					for (int i = 0; i < hww; i++)
					{
							tmp_global_x = global_x - i;

							tmp_local_pos = local_pos - i;
							if (tmp_global_x < 0)
							{
									tile1[tmp_local_pos] = tile2[tmp_local_pos] = 0;
									//? Ne znam dali e ova potrebno ama better safe than sorry
							}
							else
							{
									tmp_global_pos = global_pos - i;
									tile1[tmp_local_pos - i] = arr1[tmp_global_pos * 3];
									tile2[tmp_local_pos - i] = arr2[tmp_global_pos * 3];
							}
					}
			}
			// Right excess
			if (local_x == 32 + 2 - 1)
			{
					for (int i = 0; i < hww; i++)
					{
							tmp_global_x = global_x + i;
							tmp_local_pos = local_pos + i;
							if (tmp_global_x >= w)
							{
									tile1[tmp_local_pos] = tile2[tmp_local_pos] = 0;
									//? Ne znam dali e ova potrebno ama better safe than sorry
							}
							else
							{
									tmp_global_pos = global_pos + i;
									tile1[tmp_local_pos - i] = arr1[tmp_global_pos * 3];
									tile2[tmp_local_pos - i] = arr2[tmp_global_pos * 3];
							}
					}
			}

			// Top excess
			if (local_y == 2)
			{
					for (int i = 0; i < hwh; i++)
					{
							tmp_global_y = global_y - i;
							tmp_local_y = local_y - i;

							tmp_local_pos = tmp_local_y * TILE_SIZE + local_x;

							if (tmp_global_y < 0)
							{
									tile1[tmp_local_pos] = tile2[tmp_local_pos] = 0;
									//? Ne znam dali e ova potrebno ama better safe than sorry
							}
							else
							{
									tmp_global_pos = tmp_global_y * w + global_x;
									tile1[tmp_local_pos - i] = arr1[tmp_global_pos * 3];
									tile2[tmp_local_pos - i] = arr2[tmp_global_pos * 3];
							}
					}
			}
			// Bottom excess
			if (local_y == 32 + 2 - 1)
			{
					for (int i = 0; i < hwh; i++)
					{
							tmp_global_y = global_y + i;
							tmp_local_y = local_y + i;

							tmp_local_pos = tmp_local_y * TILE_SIZE + local_x;

							if (tmp_global_y >= h)
							{
									tile1[tmp_local_pos] = tile2[tmp_local_pos] = 0;
									//? Ne znam dali e ova potrebno ama better safe than sorry
							}
							else
							{
									tmp_global_pos = tmp_global_y * w + global_x;
									tile1[tmp_local_pos - i] = arr1[tmp_global_pos * 3];
									tile2[tmp_local_pos - i] = arr2[tmp_global_pos * 3];
							}
					}
			}
			// Loading finished

			__syncthreads();

			int start_local_x = local_x - hwh;
			int start_local_y = local_y - hwh;

			int tmp = 0;
			for (int i = 0; i < wh; i++)
			{
					tmp_local_y = start_local_y + i;
					if (tmp_local_y < 0 || tmp_local_y >= TILE_SIZE_Y)
					{
							continue;
					}
					for (int j = 0; j < ww; j++)
					{
							tmp_local_x = start_local_x + j;
							if (tmp_local_x < 0 || tmp_local_x >= TILE_SIZE_X)
							{
									continue;
							}
							tmp_local_pos = tmp_local_y * TILE_SIZE_X + tmp_local_x;
							tmp += arr1[tmp_local_pos] * arr2[tmp_local_pos];
					}
			}

			dest[global_pos] = tmp;
	}

	__global__ void g_srm_3ch_1ch(unsigned char *arr1, unsigned char *arr2, int *dest, int w, int h, int ww, int wh)
	{

			int x = threadIdx.x + blockIdx.x * blockDim.x;
			int y = threadIdx.y + blockIdx.y * blockDim.y;

			int pos = y * w + x;

			int start_x = x - (ww >> 1);
			int start_y = y - (wh >> 1);

			int tmp_pos, tmp_x, tmp_y;
			int tmp = 0;

			for (int i = 0; i < wh; i++)
			{
					tmp_y = start_y + i;
					if (tmp_y < 0 || tmp_y >= h)
					{
							continue;
					}
					for (int j = 0; j < ww; j++)
					{
							tmp_x = start_x + j;
							if (tmp_x < 0 || tmp_x >= w)
							{
									continue;
							}
							tmp_pos = tmp_y * w + tmp_x;
							tmp += arr1[tmp_pos * 3] * arr2[tmp_pos * 3];
					}
			}

			dest[pos] = tmp;
	}

	/// @brief CUDA kernel to multiply arr1[i] * arr2[i] for all i belonging to a window around each point of the matrices
	/// @param arr1 Matrix1
	/// @param arr2 Matrix2
	/// @param w Width
	/// @param h Height
	/// @param ww Window Width
	/// @param wh Window Height
	/// @param dest Destination Matrix
	__global__ void g_srm_1ch(const unsigned char *arr1, const unsigned char *arr2, int w, int h, int ww, int wh, int *dest)
	{

			int x = threadIdx.x + blockIdx.x * blockDim.x;
			int y = threadIdx.y + blockIdx.y * blockDim.y;

			if (x >= w || y >= h)
			{
					return;
			}

			int tmp_pos, tmp_x, tmp_y, pos, start_x, start_y;
			int tmp = 0;

			int hww = ww >> 1;
			int hwh = wh >> 1;
			pos = y * w + x;
			start_x = x - hww;
			start_y = y - hwh;

			for (int p = 0; p < wh; p++)
			{
					tmp_y = start_y + p;
					if (tmp_y < 0 || tmp_y >= h)
					{
							continue;
					}
					for (int q = 0; q < ww; q++)
					{
							tmp_x = start_x + q;
							if (tmp_x < 0 || tmp_x >= w)
							{
									continue;
							}
							tmp_pos = tmp_y * w + tmp_x;
							tmp += arr1[tmp_pos] * arr2[tmp_pos];
					}
			}
			dest[pos] = tmp;
	}

	/// @brief Launches a CUDA kernel to multiply arr1[i] * arr2[i] for all i belonging to a window around each point of the matrices
	/// @param arr1_h Matrix1
	/// @param arr2_h Matrix2
	/// @param w Width
	/// @param h Height
	/// @param ww Window Width
	/// @param wh Window Height
	/// @param dest_h Destination Matrix
	void srm_1ch(const unsigned char *arr1_h, const unsigned char *arr2_h, int w, int h, int ww, int wh, int *dest_h)
	{

			unsigned char *arr1_d;
			unsigned char *arr2_d;
			int *dest_d;

			hipMalloc((void **)&arr1_d, w * h * sizeof(unsigned char));
			hipMalloc((void **)&arr2_d, w * h * sizeof(unsigned char));
			hipMalloc((void **)&dest_d, w * h * sizeof(int));

			hipMemcpy(arr1_d, arr1_h, w * h * sizeof(unsigned char), hipMemcpyHostToDevice);
			hipMemcpy(arr2_d, arr2_h, w * h * sizeof(unsigned char), hipMemcpyHostToDevice);

			int NUM_OF_THREADS = 32;
			dim3 blockSize(NUM_OF_THREADS, NUM_OF_THREADS);
			int GRID_SIZE_X = (int)ceil((float)w / (float)NUM_OF_THREADS);
			int GRID_SIZE_Y = (int)ceil((float)h / (float)NUM_OF_THREADS);
			dim3 gridSize(GRID_SIZE_X, GRID_SIZE_Y);

			g_srm_1ch<<<blockSize, gridSize>>>(arr1_d, arr2_d, w, h, ww, wh, dest_d);

			hipMemcpy(dest_h, dest_d, w * h * sizeof(int), hipMemcpyDeviceToHost);

			hipFree(arr1_d);
			hipFree(arr2_d);
			hipFree(dest_d);
	}

	/// @brief CUDA kernel to multiply arr1[i] * arr2[i] for all i belonging to a window around each point of the matrices
	/// @param arr1 Matrix1
	/// @param arr2 Matrix2
	/// @param w Width
	/// @param h Height
	/// @param ww Window Width
	/// @param wh Window Height
	/// @param dest Destination Matrix
	__global__ void g_srm_1ch_float(const float *arr1, const float *arr2, int w, int h, int ww, int wh, float *dest)
	{

			int x = threadIdx.x + blockIdx.x * blockDim.x;
			int y = threadIdx.y + blockIdx.y * blockDim.y;

			if (x >= w || y >= h)
			{
					return;
			}

			int tmp_pos, tmp_x, tmp_y, pos, start_x, start_y;
			float tmp = 0;

			int hww = ww >> 1;
			int hwh = wh >> 1;
			pos = y * w + x;
			start_x = x - hww;
			start_y = y - hwh;

			for (int p = 0; p < wh; p++)
			{
					tmp_y = start_y + p;
					if (tmp_y < 0 || tmp_y >= h)
					{
							continue;
					}
					for (int q = 0; q < ww; q++)
					{
							tmp_x = start_x + q;
							if (tmp_x < 0 || tmp_x >= w)
							{
									continue;
							}
							tmp_pos = tmp_y * w + tmp_x;
							tmp += arr1[tmp_pos] * arr2[tmp_pos];
					}
			}
			dest[pos] = tmp;
	}

	/// @brief Launches a CUDA kernel to multiply arr1[i] * arr2[i] for all i belonging to a window around each point of the matrices
	/// @param arr1_h Matrix1
	/// @param arr2_h Matrix2
	/// @param w Width
	/// @param h Height
	/// @param ww Window Width
	/// @param wh Window Height
	void srm_1ch_float(const float *arr1_h, const float *arr2_h, int w, int h, int ww, int wh, float *dest_h)
	{

		float *arr1_d;
		float *arr2_d;
		float *dest_d;

		hipMalloc((void **)&arr1_d, w * h * sizeof(float));
		hipMalloc((void **)&arr2_d, w * h * sizeof(float));
		hipMalloc((void **)&dest_d, w * h * sizeof(float));

		hipMemcpy(arr1_d, arr1_h, w * h * sizeof(float), hipMemcpyHostToDevice);
		hipMemcpy(arr2_d, arr2_h, w * h * sizeof(float), hipMemcpyHostToDevice);

		int NUM_OF_THREADS = 32;
		dim3 blockSize(NUM_OF_THREADS, NUM_OF_THREADS);
		int GRID_SIZE_X = (int)ceil((float)w / (float)NUM_OF_THREADS);
		int GRID_SIZE_Y = (int)ceil((float)h / (float)NUM_OF_THREADS);
		dim3 gridSize(GRID_SIZE_X, GRID_SIZE_Y);

		// sumReductionAndMultOverWindowGPU1CH_Tiled<<<blockSize, gridSize>>>(arr1_d, arr2_d, w, h, ww, wh, dest_d);
		g_srm_1ch_float<<<blockSize, gridSize>>>(arr1_d, arr2_d, w, h, ww, wh, dest_d);

		hipMemcpy(dest_h, dest_d, w * h * sizeof(float), hipMemcpyDeviceToHost);

		hipFree(arr1_d);
		hipFree(arr2_d);
		hipFree(dest_d);
	}

	//? Optimization Notes:
	__global__ void g_srm_1ch_tiled(const unsigned char *arr1, const unsigned char *arr2, int w, int h, int ww, int wh, int *dest, int TILE_SIZE_X, int TILE_SIZE_Y)
	{

			extern __shared__ unsigned char shmem[];
			unsigned char* tile1 = shmem;
			unsigned char* tile2 = shmem + w * h;

			int hwh = wh >> 1;
			int hww = ww >> 1;

			int x_o = threadIdx.x + blockIdx.x * TILE_SIZE_X;
			int y_o = threadIdx.y + blockIdx.y * TILE_SIZE_Y;
			int pos_o = x_o + y_o * w;

			int x_i = x_o - hww;
			int y_i = y_o - hwh;
			int tile_pos = threadIdx.x + threadIdx.y * blockDim.x;
			if(x_i < 0 || x_i >= w || y_i < 0 || y_i >= h){
					tile1[tile_pos] = tile2[tile_pos] = 0;
			}else{
					int pos_i = x_i + y_i * w;
					tile1[tile_pos] = arr1[pos_i];
					tile2[tile_pos] = arr2[pos_i];
			}
			// Loading finished

			__syncthreads();

			if(x_o >= w || y_o >= h){
					return;
			}
			if(threadIdx.x >= TILE_SIZE_X || threadIdx.y >= TILE_SIZE_Y){
					return; 
			}

			int tmp = 0;
			int tmp_x, tmp_y, tmp_pos;
			for (int i = 0; i < wh; i++)
			{
					tmp_y = threadIdx.y + i;
					for (int j = 0; j < ww; j++)
					{
							tmp_x = threadIdx.x + j;
							tmp_pos = tmp_x + tmp_y * blockDim.x; 
							tmp += tile1[tmp_pos] * tile2[tmp_pos];
					}
			}

			dest[pos_o] = tmp;
	}

	/// @brief Launches a CUDA kernel to multiply arr1[i] * arr2[i] for all i belonging to a window around each point of the matrices
	/// @param arr1_h Matrix1
	/// @param arr2_h Matrix2
	/// @param w Width
	/// @param h Height
	/// @param ww Window Width
	/// @param wh Window Height
	/// @param dest_h Destination Matrix
	void srm_1ch_tiled(const unsigned char *arr1_h, const unsigned char *arr2_h, int w, int h, int ww, int wh, int *dest_h)
	{

			unsigned char *arr1_d;
			unsigned char *arr2_d;
			int *dest_d;

			hipMalloc((void **)&arr1_d, w * h * sizeof(unsigned char));
			hipMalloc((void **)&arr2_d, w * h * sizeof(unsigned char));
			hipMalloc((void **)&dest_d, w * h * sizeof(int));

			hipMemcpy(arr1_d, arr1_h, w * h * sizeof(unsigned char), hipMemcpyHostToDevice);
			hipMemcpy(arr2_d, arr2_h, w * h * sizeof(unsigned char), hipMemcpyHostToDevice);

			int NUM_OF_THREADS = 32;
			int TILE_SIZE_X = NUM_OF_THREADS - ww + 1;
			int TILE_SIZE_Y = NUM_OF_THREADS - wh + 1;
			dim3 blockSize(NUM_OF_THREADS, NUM_OF_THREADS);
			int GRID_SIZE_X = (int)ceil((float)w / (float)TILE_SIZE_X);
			int GRID_SIZE_Y = (int)ceil((float)h / (float)TILE_SIZE_Y);
			dim3 gridSize(GRID_SIZE_X, GRID_SIZE_Y);

			g_srm_1ch_tiled<<<blockSize, gridSize, w * h * sizeof(unsigned char) * 2>>>(arr1_d, arr2_d, w, h, ww, wh, dest_d, TILE_SIZE_X, TILE_SIZE_Y);

			hipMemcpy(dest_h, dest_d, w * h * sizeof(int), hipMemcpyDeviceToHost);

			hipFree(arr1_d);
			hipFree(arr2_d);
			hipFree(dest_d);
	}

	/// @brief CUDA kernel for solving the inverse matrix and calculating optical flow
	/// @param sumIx2
	/// @param sumIy2
	/// @param sumIxIy
	/// @param sumIxIt
	/// @param sumIyIt
	/// @param optFlow float* of the destination of the optical flow calculation
	/// @param w Width
	/// @param h Height
	__global__ void g_inv_matrix(int *sumIx2, int *sumIy2, int *sumIxIy, int *sumIxIt, int *sumIyIt, float *optFlow, int w, int h)
	{

			int x = threadIdx.x + blockDim.x * blockIdx.x;
			int y = threadIdx.y + blockDim.y * blockIdx.y;

			if (x >= w || y >= h)
			{
					return;
			}

			int pos = y * w + x;

			double a, b, c, d;
			a = (double)sumIx2[pos];
			b = c = (double)sumIxIy[pos];
			d = (double)sumIy2[pos];
			double prefix = 1 / (a * d - b * c);
			a *= prefix;
			b *= prefix;
			c *= prefix;
			d *= prefix;

			float u = -d * sumIxIt[pos] + b * sumIyIt[pos];
			float v = c * sumIxIt[pos] - a * sumIyIt[pos];

			optFlow[pos * 2] = u;
			optFlow[pos * 2 + 1] = v;
	}

	/// @brief Solves the inverse matrix in the optical flow equation and calculates the opticalFlow
	/// @param sumIx2
	/// @param sumIy2
	/// @param sumIxIy
	/// @param sumIxIt
	/// @param sumIyIt
	/// @param optFlowPyramid Optical Flow Pyramid
	/// @param level Current Level of the pyramid
	/// @param w Current Width
	/// @param h Current Height
	void inverse_matrix(int *sumIx2, int *sumIy2, int *sumIxIy, int *sumIxIt, int *sumIyIt, float **optFlowPyramid, int level, int w, int h)
	{
			int *sumIx2_d;
			int *sumIy2_d;
			int *sumIxIy_d;
			int *sumIxIt_d;
			int *sumIyIt_d;
			float *optFlow_d;

			size_t size = w * h * sizeof(int);
			hipMalloc((void **)&sumIx2_d, size);
			hipMalloc((void **)&sumIy2_d, size);
			hipMalloc((void **)&sumIxIy_d, size);
			hipMalloc((void **)&sumIxIt_d, size);
			hipMalloc((void **)&sumIyIt_d, size);

			hipMemcpy(sumIx2_d, sumIx2, size, hipMemcpyHostToDevice);
			hipMemcpy(sumIy2_d, sumIy2, size, hipMemcpyHostToDevice);
			hipMemcpy(sumIxIy_d, sumIxIy, size, hipMemcpyHostToDevice);
			hipMemcpy(sumIxIt_d, sumIxIt, size, hipMemcpyHostToDevice);
			hipMemcpy(sumIyIt_d, sumIyIt, size, hipMemcpyHostToDevice);

			size_t flowSize = w * h * 2 * sizeof(float);
			hipMalloc((void **)&optFlow_d, flowSize);

			int NUM_OF_THREADS = 32;
			dim3 blockSize(NUM_OF_THREADS, NUM_OF_THREADS);
			int GRID_SIZE_X = (int)ceil(w / NUM_OF_THREADS);
			int GRID_SIZE_Y = (int)ceil(h / NUM_OF_THREADS);
			dim3 gridSize(GRID_SIZE_X, GRID_SIZE_Y);
			g_inv_matrix<<<blockSize, gridSize>>>(sumIx2_d, sumIy2_d, sumIxIy_d, sumIxIt_d, sumIyIt_d, optFlow_d, w, h);

			hipMemcpy(optFlowPyramid[level], optFlow_d, flowSize, hipMemcpyDeviceToHost);

			hipFree(sumIx2_d);
			hipFree(sumIy2_d);
			hipFree(sumIxIy_d);
			hipFree(sumIxIt_d);
			hipFree(sumIyIt_d);

			hipFree(optFlow_d);
	}

	/// @brief CUDA kernel for solving the inverse matrix and calculating optical flow
	/// @param sumIx2
	/// @param sumIy2
	/// @param sumIxIy
	/// @param sumIxIt
	/// @param sumIyIt
	/// @param optFlow float* of the destination of the optical flow calculation
	/// @param w Width
	/// @param h Height
	__global__ void g_inv_matrix_float(float *sumIx2, float *sumIy2, float *sumIxIy, float *sumIxIt, float *sumIyIt, float *optFlow, int w, int h)
	{
			int x = threadIdx.x + blockDim.x * blockIdx.x;
			int y = threadIdx.y + blockDim.y * blockIdx.y;

			if (x >= w || y >= h)
			{
					return;
			}

			int pos = y * w + x;

			double a, b, c, d;
			a = (double)sumIx2[pos];
			b = c = (double)sumIxIy[pos];
			d = (double)sumIy2[pos];
			double prefix = 1 / (a * d - b * c);
			a *= prefix;
			b *= prefix;
			c *= prefix;
			d *= prefix;

			float u = -d * sumIxIt[pos] + b * sumIyIt[pos];
			float v = c * sumIxIt[pos] - a * sumIyIt[pos];

			optFlow[pos * 2] = u;
			optFlow[pos * 2 + 1] = v;
	}

	/// @brief Solves the inverse matrix in the optical flow equation and calculates the opticalFlow
	/// @param sumIx2
	/// @param sumIy2
	/// @param sumIxIy
	/// @param sumIxIt
	/// @param sumIyIt
	/// @param optFlowPyramid Optical Flow Pyramid
	/// @param level Current Level of the pyramid
	/// @param w Current Width
	/// @param h Current Height
	void inverse_matrix_float(float *sumIx2, float *sumIy2, float *sumIxIy, float *sumIxIt, float *sumIyIt, float **optFlowPyramid, int level, int w, int h)
	{
			float *sumIx2_d;
			float *sumIy2_d;
			float *sumIxIy_d;
			float *sumIxIt_d;
			float *sumIyIt_d;
			float *optFlow_d;

			size_t size = w * h * sizeof(float);
			hipMalloc((void **)&sumIx2_d, size);
			hipMalloc((void **)&sumIy2_d, size);
			hipMalloc((void **)&sumIxIy_d, size);
			hipMalloc((void **)&sumIxIt_d, size);
			hipMalloc((void **)&sumIyIt_d, size);

			hipMemcpy(sumIx2_d, sumIx2, size, hipMemcpyHostToDevice);
			hipMemcpy(sumIy2_d, sumIy2, size, hipMemcpyHostToDevice);
			hipMemcpy(sumIxIy_d, sumIxIy, size, hipMemcpyHostToDevice);
			hipMemcpy(sumIxIt_d, sumIxIt, size, hipMemcpyHostToDevice);
			hipMemcpy(sumIyIt_d, sumIyIt, size, hipMemcpyHostToDevice);

			size_t flowSize = w * h * 2 * sizeof(float);
			hipMalloc((void **)&optFlow_d, flowSize);

			int NUM_OF_THREADS = 32;
			dim3 blockSize(NUM_OF_THREADS, NUM_OF_THREADS);
			int GRID_SIZE_X = (int)ceil(w / NUM_OF_THREADS);
			int GRID_SIZE_Y = (int)ceil(h / NUM_OF_THREADS);
			dim3 gridSize(GRID_SIZE_X, GRID_SIZE_Y);
			g_inv_matrix_float<<<blockSize, gridSize>>>(sumIx2_d, sumIy2_d, sumIxIy_d, sumIxIt_d, sumIyIt_d, optFlow_d, w, h);

			hipMemcpy(optFlowPyramid[level], optFlow_d, flowSize, hipMemcpyDeviceToHost);

			hipFree(sumIx2_d);
			hipFree(sumIy2_d);
			hipFree(sumIxIy_d);
			hipFree(sumIxIt_d);
			hipFree(sumIyIt_d);

			hipFree(optFlow_d);
	}

	/// @brief A function that calculates optical flow for a single level of the Gaussian Pyramid using GPU functions
	/// @param prev Previous Image
	/// @param next Next Image
	/// @param w Image Width at this level
	/// @param h Image Height at this level
	/// @param optFlowPyramid An array containing the optical flow field at every level of the pyramid
	/// @param level Level of the Gaussian pyramid
	/// @param maxLevel MaxLevel of the Gaussian pyramid
	void calc_opt_flow(const unsigned char *prev, unsigned char *next, int w, int h, float **optFlowPyramid, int level, int maxLevel)
	{
			// optFlowPyramid is the pyramid of all optical flows
			// optFlowPyramid[i] is the optical flow field, described by a vector (u, v) at each point

			// STEP 0
			// SHIFT NEXT IMAGE BACK BY PREVIOUSLY CALCULATED OPTICAL FLOW
			// Ova se pravi za celiot dosega presmetan optical flow
			unsigned char *shifted = (unsigned char *)malloc(w * h * 3 * sizeof(unsigned char));
			if (level != maxLevel - 1)
			{
					cpu::shift_back_pyramid(next, w, h, level, maxLevel, optFlowPyramid, shifted);
					next = shifted;
			}

			//!Ke vidime dali ova ke raboti
			//!RABOTI!!!!!!!!!
			// STEP 1
			// calculate partial derivatives at all points using kernels for finite differences (Ix, Iy, It)

			float *Ix = (float *)malloc(w * h * sizeof(float));
			gpu::conv_3ch_1ch_tiled_uchar_float(prev, w, h, Ix, Dx_3x3, 3, 3);

			float *Iy = (float *)malloc(w * h * sizeof(float));
			gpu::conv_3ch_1ch_tiled_uchar_float(prev, w, h, Iy, Dy_3x3, 3, 3);

			float *It1 = (float *)malloc(w * h * sizeof(float));
			gpu::conv_3ch_1ch_tiled_uchar_float(prev, w, h, It1, Dt_3x3, 3, 3);
			float *It2 = (float *)malloc(w * h * sizeof(float));
			gpu::conv_3ch_1ch_tiled_uchar_float(next, w, h, It2, Dt_3x3, 3, 3);
			float *It = It1; // ova za da bide podobro optimizirano
			utils::arr_sub_float(It2, It1, w * h, It);

			// STEP 2
			// Calculate sumIx2, sumIy2, sumIxIy, sumIxIt, sumIyIt
			int ww = 9;
			int wh = 9;
			
			float *sumIx2 = (float *)malloc(w * h * sizeof(float));
			gpu::srm_1ch_float(Ix, Ix, w, h, ww, wh, sumIx2);

			float *sumIy2 = (float *)malloc(w * h * sizeof(float));
			gpu::srm_1ch_float(Iy, Iy, w, h, ww, wh, sumIy2);

			float *sumIxIy = (float *)malloc(w * h * sizeof(float));
			gpu::srm_1ch_float(Ix, Iy, w, h, ww, wh, sumIxIy);

			float *sumIxIt = (float *)malloc(w * h * sizeof(float));
			gpu::srm_1ch_float(Ix, It, w, h, ww, wh, sumIxIt);

			float *sumIyIt = (float *)malloc(w * h * sizeof(int));
			gpu::srm_1ch_float(Iy, It, w, h, ww, wh, sumIyIt);

			// STEP 3
			// Calculate the optical flow vector at every point (i, j)
			gpu::inverse_matrix_float(sumIx2, sumIy2, sumIxIy, sumIxIt, sumIyIt, optFlowPyramid, level, w, h);

			// Free all malloc memory
			free(Ix);
			free(Iy);
			free(It1);
			free(It2);

			free(sumIx2);
			free(sumIy2);
			free(sumIxIy);
			free(sumIxIt);
			free(sumIyIt);

			free(shifted);
	}

	//ova ima prostor do 10x10
	__constant__ double gaus_kernel_10x10_gpu[100];

	__global__ void g_bilinear_filter(unsigned char *src, unsigned char *gray, unsigned char *dest, int w, int h, int ww, int wh, double sigmaB)
	{
			int x = threadIdx.x + blockIdx.x * blockDim.x;
			int y = threadIdx.y + blockIdx.y * blockDim.y;
			
			if(x < 0 || y < 0 || x >= w || y >= h){
					return;
			}

			int hwh = wh >> 1;
			int hww = ww >> 1;

			const 
			double* gaus_mask = gaus_kernel_10x10_gpu;

			int pos = y * w + x;
			double wsb = 0;

			int start_y = y - hwh;
			int start_x = x - hww;

			double f_ij = gray[pos * 3];

			double tmp[3] = {0, 0, 0};
			for (int m = 0; m < wh; m++)
			{
					int c_y = start_y + m;
					if (c_y < 0 || c_y >= h)
					{
							continue;
					}
					for (int n = 0; n < ww; n++)
					{
							double sigmaB2 = sigmaB * sigmaB;

							int c_x = start_x + n;

							if (c_x < 0 || c_x >= w)
							{
									continue;
							}

							int c_pos = c_y * w + c_x;

							double f_mn = gray[c_pos * 3];
							double k = f_mn - f_ij;
							double k2 = k * k;

							double n_b = 1.0 / (2.0 * M_PI * sigmaB2) * pow(M_E, -0.5 * (k2) / sigmaB2);
							double n_s = gaus_mask[m * ww + n];

							wsb += n_b * n_s;
							tmp[0] += src[c_pos * 3] * n_b * n_s;
							tmp[1] += src[c_pos * 3 + 1] * n_b * n_s;
							tmp[2] += src[c_pos * 3 + 2] * n_b * n_s;
					}
			}
			tmp[0] /= wsb;
			tmp[1] /= wsb;
			tmp[2] /= wsb;

			dest[pos * 3] = (unsigned char)tmp[0];
			dest[pos * 3 + 1] = (unsigned char)tmp[1];
			dest[pos * 3 + 2] = (unsigned char)tmp[2];
	}

	void bilinear_filter(unsigned char *src, unsigned char *gray, unsigned char *dest, int w, int h, int ww, int wh, double sigmaS, double sigmaB)
	{
			double* gaus_mask = (double*) malloc(ww * wh * sizeof(double)); 
			utils::generate_gaussian_kernel(sigmaS, ww, gaus_mask);

			unsigned char* src_d;
			unsigned char* gray_d;
			unsigned char* dest_d;
			
			hipMalloc((void**) &src_d, w * h * 3 * sizeof(unsigned char));
			hipMalloc((void**) &gray_d, w * h * 3 * sizeof(unsigned char));
			hipMalloc((void**) &dest_d, w * h * 3 * sizeof(unsigned char));

			hipMemcpyToSymbol(HIP_SYMBOL(gaus_kernel_10x10_gpu), gaus_mask, ww * wh * sizeof(double));

			hipMemcpy(src_d, src, w * h * 3 * sizeof(unsigned char), hipMemcpyHostToDevice);
			hipMemcpy(gray_d, gray, w * h * 3 * sizeof(unsigned char), hipMemcpyHostToDevice);

			int NUM_OF_THREADS = 32;
			dim3 blockSize(NUM_OF_THREADS, NUM_OF_THREADS);
			int GRID_SIZE_X = (int) ceil((float) w / (float) NUM_OF_THREADS);
			int GRID_SIZE_Y = (int) ceil((float) h / (float) NUM_OF_THREADS);
			dim3 gridSize(GRID_SIZE_X, GRID_SIZE_Y);

			g_bilinear_filter<<<blockSize, gridSize>>>(src_d, gray_d, dest_d, w, h, ww, wh, sigmaB);

			hipMemcpy(dest, dest_d, w * h * 3 * sizeof(unsigned char), hipMemcpyDeviceToHost);

			hipFree(src_d);
			hipFree(gray_d);
			hipFree(dest_d);

			free(gaus_mask);
	}

}
